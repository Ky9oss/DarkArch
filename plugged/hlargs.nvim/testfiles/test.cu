
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>

class MyClass {
    int arg0_;
    float* arg1;
    MyClass(int arg0, float* arg1): arg0_(arg0){
        this->arg1 = arg1;
    }

    int fn(char* arg2, int arg3 = 0){
        return arg3 + arg2[0];
    }
};

struct mystr {
    int argc;
};

template<typename T>
auto fn(T arg) {
    return std::vector<T>({arg});
}

__host__ __device__ float dev(float arg0) {
    return arg0 + 1.f;
}

template<typename T>
__global__ void kernel(const T* arg0, float* arg1, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;

    arg1[i] = dev(static_cast<T>(arg0[i]));
}

int main(int argc, char* argv[]){

    std::cout<<argv[argc-1]<<std::endl;

    std::vector<int> vec;
    struct mystr a;
    a.argc = 0;

    try {
        sort(vec.begin(), vec.end(), [argc](const int & a, const int & b) -> bool {
            int c = b - argc;
            return a > c;
        });
    } catch (const std::exception& e) {
        std::cout << e.what();
    }
    
    int* input;
    float* result;
    int N = 256;
    kernel<int><<<dim3(16), dim3(16)>>>(input, result, N);

    hipDeviceReset();
    return 0;
}
